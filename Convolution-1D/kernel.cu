#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void convolution_global_memory(float *N, float *M, float *P, int Width,int MASK_WIDTH) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float Pvalue = 0;

	int n_start_point = i - (MASK_WIDTH / 2);

	for (int j = 0; j < MASK_WIDTH; j++) {
		if (n_start_point + j >= 0 && n_start_point + j < Width) {
			Pvalue += N[n_start_point + j] * M[j];
		}
	}

	P[i] = Pvalue;
	
}
int main()
{
	float *h_N, *h_M, *h_P;
	float *d_N, *d_M, *d_P;
	int N, M;
	hipError_t cudaStatus;
	printf("Enter the Array Size");
	scanf_s("%d", &N);
	
	printf("Enter the Mask Size");
	scanf_s("%d", &M);
	
	h_N = (float*)malloc(sizeof(float) * N);
	printf("Enter the Array");
	for (int k = 0; k < N; k++)
	{
		scanf("%f",&h_N[k]);
	}
	h_P = (float*)malloc(sizeof(float) * N);
	h_M = (float*)malloc(sizeof(float) * M);
	printf("Enter the Mask");
	for (int k = 0; k < M; k++)
	{
		scanf("%f",&h_M[k]);
	}
	cudaStatus=hipMalloc((void**)&d_N, sizeof(float) * N);
	printf("%d", cudaStatus);
	cudaStatus=hipMalloc((void**)&d_P, sizeof(float) * N);
	printf("%d", cudaStatus);
	cudaStatus=hipMalloc((void**)&d_M, sizeof(float) * M);
	printf("%d", cudaStatus);
	cudaStatus=hipMemcpy(d_N, h_N, N * sizeof(float), hipMemcpyHostToDevice);
	printf("%d", cudaStatus);
	cudaStatus=hipMemcpy(d_M, h_M, M * sizeof(float), hipMemcpyHostToDevice);
	printf("%d\n", cudaStatus);
	convolution_global_memory <<< 1, N >>>(d_N, d_M, d_P, N, M);
	cudaStatus=hipMemcpy(h_P, d_P, N * sizeof(float), hipMemcpyDeviceToHost);
	printf("%d", cudaStatus);
	printf("\nResult is\n");
		for (int k = 0; k < N; k++)
		{
			printf("%f\t", h_P[k]);
		}
	hipFree(d_N);
	hipFree(d_M);
	hipFree(d_P);
	return 0;
}
