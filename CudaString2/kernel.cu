#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void E7(char* A, char* C)
{
	int idx = threadIdx.x;
	int size = blockDim.x;
	int i = 0;
	while (i <= idx)
	{
		C[i + (idx * (idx + 1) / 2)] = A[idx];
		i++;

	}
}
int main()
{
	char h_A[5] = "PCAP";

	char h_C[11];
	char* d_A, * d_C;
	hipMalloc(&d_A, sizeof(char) * 5);

	hipMalloc(&d_C, sizeof(char) * 11);
	hipMemcpy(d_A, h_A, sizeof(char) * 5, hipMemcpyHostToDevice);

	E7 << <1, 4 >> > (d_A, d_C);
	hipMemcpy(h_C, d_C, sizeof(char) * 11, hipMemcpyDeviceToHost);
	for (int i = 0; i < 11; i++)
	{
		printf("%c", h_C[i]);
	}
	hipFree(d_A);

	hipFree(d_C);
	return 0;
}