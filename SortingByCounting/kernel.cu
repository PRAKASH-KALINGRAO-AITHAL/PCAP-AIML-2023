#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
__global__ void SortP(long int* a, long int* c)
{
	int pos,data,j,temp;
	int i = threadIdx.x;
	int n = blockDim.x;

		data = a[i]; 
		pos = 0;
		for (j = 0; j < n; j++)
			if (a[j] < data || (a[j] == data && j < i))
				pos++;
		c[pos] = data;

}


int main()
{
	long int* dev_a = 0;
	long int* dev_c = 0;
	long int* a;
	long int* c;
	long int size;
	hipError_t cudaStatus;
	printf("Enter the size of the array");
	scanf_s("%ld", &size);
	a = (long int*)malloc(sizeof(long int) * size);
	c = (long int*)malloc(sizeof(long int) * size);
	printf("Enter the array");
	for (long int i = 0; i < size; i++)
	{
		scanf_s("%d", &a[i]);
	}
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(long int));
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(long int));
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(long int), hipMemcpyHostToDevice);

	

		SortP << < 1, size >> > (dev_a,dev_c);
		

	
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(long int), hipMemcpyDeviceToHost);
	printf("Result\n");
	for (long int w = 0; w < size; w++)
	{
		printf("%ld\t", c[w]);
	}
	hipFree(dev_a);
	hipFree(dev_c);
	return 0;
}
