#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
__global__ void vecAdd(int* A, int* B, int* C, int n)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < n)
	{
		C[idx] = A[idx] + B[idx];
	}

}
int main()
{
	int h_A[8] = { 1,1,1,1,1,1,1,1 };
	int h_B[8] = { 1,1,1,1,1,1,1,1 };
	int h_C[8];
	int* d_A, * d_B, * d_C;
	hipMalloc(&d_A, sizeof(int) * 8);
	hipMalloc(&d_B, sizeof(int) * 8);
	hipMalloc(&d_C, sizeof(int) * 8);
	hipMemcpy(d_A, h_A, sizeof(int) * 8, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(int) * 8, hipMemcpyHostToDevice);
	vecAdd << <1, 32 >> > ( d_A,d_B,d_C, 8);
	hipMemcpy(h_C, d_C, sizeof(int) * 8, hipMemcpyDeviceToHost);
	for (int i = 0; i < 8; i++)
	{
		printf("%d", h_C[i]);
	}
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}
