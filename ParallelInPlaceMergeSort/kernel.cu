#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<stdlib.h>
#include<math.h>
__global__ void CompareExchange(long int* A, int pass)
{
	int idx = threadIdx.x;
	if (idx <= pass)
	{
		int i = idx * 2 * pass;
		int j = idx * 2 * pass + pass;
		int flag = 0;
		int z = 0;

		for (int k = 0; k <= pass; k += 1)
		{
			

			flag = 0;
			for (z = 0; z < pass; z += 1)
			{
				
				if (A[i + k] >= A[j + z])
				{
					int temp = A[i + k];
					A[i + k] = A[j + z];
					A[j + z] = temp;
					flag++;
					k++;
				}
				if (A[i + k] < A[j + z])
				{
					flag++;
					k++;
				}

			}

			while (flag > 0)
			{
				k--;
				flag--;
			}

		}
		
	}
}
int main()
{
    long int* dev_a = 0;
    long int* a;
    long int size;
    hipError_t cudaStatus;
    printf("Enter the size of the array");
    scanf_s("%ld", &size);
    a = (long int*)malloc(sizeof(long int) * size);
    printf("Enter the array");
    for (int i = 0; i < size; i++)
    {
        scanf("%d", &a[i]);
    }

    long int N = size / 2;
	long int i = 1;
	while (N >= 1)
	{
		cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(long int));
		cudaStatus = hipMemcpy(dev_a, a, size * sizeof(long int), hipMemcpyHostToDevice);
		CompareExchange <<< 1,N  >> > (dev_a, i);
		cudaStatus = hipMemcpy(a, dev_a, size * sizeof(long int), hipMemcpyDeviceToHost);
		i = i * 2;
		N = N / 2;
	}
    printf("Result\n");
    for (int w = 0; w < size; w++)
    {
        printf("%d\t",a[w]);
    }
    hipFree(dev_a);
    return 0;
}


