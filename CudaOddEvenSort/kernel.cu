#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
__global__ void odd(long int* A)
{
	int idx = threadIdx.x;
	int size = blockDim.x;
	if ((idx % 2) != 0 && idx + 1 <= size - 1)
	{

		if (A[idx] > A[idx + 1])
		{
			int temp = A[idx];
			A[idx] = A[idx + 1];
			A[idx + 1] = temp;
		}

	}
}
__global__ void even(long int* A)
{
	int idx = threadIdx.x;
	int size = blockDim.x;
	if ((idx % 2) == 0 && idx <= size - 1)
	{

		if (A[idx] > A[idx + 1])
		{
			int temp = A[idx];
			A[idx] = A[idx + 1];
			A[idx + 1] = temp;
		}
	}
}
int main()
{
	long int* dev_a = 0;
	long int* a;
	long int size;
	hipError_t cudaStatus;
	printf("Enter the size of the array");
	scanf_s("%ld", &size);
	a = (long int*)malloc(sizeof(long int) * size);
	printf("Enter the array");
	for (int i = 0; i < size; i++)
	{
		scanf("%d", &a[i]);
	}
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(long int));
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(long int), hipMemcpyHostToDevice);

	for(int i=0;i<size/2;i++)
	{
		
		odd << < 1, size >> > (dev_a);
		even << <1, size >> > (dev_a);
		
	}
	cudaStatus = hipMemcpy(a, dev_a, size * sizeof(long int), hipMemcpyDeviceToHost);
	printf("Result\n");
	for (int w = 0; w < size; w++)
	{
		printf("%d\t", a[w]);
	}
	hipFree(dev_a);
	return 0;
}


