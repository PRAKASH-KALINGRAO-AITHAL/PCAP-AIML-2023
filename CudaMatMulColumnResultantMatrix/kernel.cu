#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#define ha 5
#define wa 4
#define hb 4
#define wb 5
#define hc 5
#define wc 5
__global__ void multiplyKernel_colwise(int* a, int* b, int* c)
{
	int cidB = threadIdx.x;
	int sum, k;
	for (int ridA = 0; ridA < ha; ridA++)
	{
		sum = 0;
		for (k = 0; k < wa; k++)
		{
			sum += (a[ridA * wa + k] * b[k * wb + cidB]);
		}
		c[ridA * wb + cidB] = sum;
	}
}



int main()
{
	int h_A[ha][wa] = { {1,2,3,1},{1,1,1,1},{1,2,2,1},{1,0,3,1},{2,2,2,1} };
	int h_B[hb][wb] = { {1,2,3,4,5},{1,1,1,1,1},{1,2,3,2,1} ,{1,1,1,1,1} };
	int h_C[hc][wc];
	int* d_A, * d_B, * d_C;
	hipMalloc(&d_A, sizeof(int) * ha * wa);
	hipMalloc(&d_B, sizeof(int) * hb * wb);
	hipMalloc(&d_C, sizeof(int) * hc * wc);
	hipMemcpy(d_A, h_A, sizeof(int) * ha * wa, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(int) * hb * wb, hipMemcpyHostToDevice);
	multiplyKernel_colwise << <1, wc >> > (d_A, d_B, d_C);

	hipMemcpy(h_C, d_C, sizeof(int) * hc * wc, hipMemcpyDeviceToHost);
	for (int i = 0; i < hc; i++)
	{
		for (int j = 0; j < wc; j++)
		{
			printf("%d\t", h_C[i][j]);
		}
		printf("\n");
	}
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}