#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define ha 8
#define wa 8
#define hb 8
#define wb 8
#define hc 8
#define wc 8
#define TILE_WIDTH 2
__global__ void multiplyKernel_elementwise(int* a, int* b, int* c)
{
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Col = bx * TILE_WIDTH + tx;
	int Row = by * TILE_WIDTH + ty;
	int Pvalue = 0;
	for (int bloc = 0; bloc < (wc / TILE_WIDTH); ++bloc)
	{
		Mds[ty][tx] = a[Row * wc + bloc * TILE_WIDTH + tx];
		Nds[ty][tx] = b[(bloc * TILE_WIDTH + ty) * wc + Col];

		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k)
		{
			Pvalue += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
	}
	c[Row * wc + Col] = Pvalue;

}



int main()
{
	int h_A[ha][wa] = { {1,2,3,1,1,2,3,1},{1,1,1,1,1,2,3,1},{1,2,2,1,1,2,3,1},{1,0,3,1,1,2,3,1},{1,2,3,1,1,2,3,1},{1,1,1,1,1,2,3,1},{1,2,2,1,1,2,3,1},{1,0,3,1,1,2,3,1} };
	int h_B[hb][wb] = { {1,2,3,4,1,2,3,1},{1,1,1,1,1,2,3,1},{1,2,3,2,1,2,3,1} ,{1,1,1,1,1,2,3,1},{1,2,3,1,1,2,3,1},{1,1,1,1,1,2,3,1},{1,2,2,1,1,2,3,1},{1,0,3,1,1,2,3,1} };
	int h_C[hc][wc];
	int* d_A, * d_B, * d_C;
	hipMalloc(&d_A, sizeof(int) * ha * wa);
	hipMalloc(&d_B, sizeof(int) * hb * wb);
	hipMalloc(&d_C, sizeof(int) * hc * wc);
	hipMemcpy(d_A, h_A, sizeof(int) * ha * wa, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(int) * hb * wb, hipMemcpyHostToDevice);
	dim3 GridDimension(wc / TILE_WIDTH, hc / TILE_WIDTH, 1);
	dim3 BlockDimension(TILE_WIDTH, TILE_WIDTH, 1);
	multiplyKernel_elementwise << < GridDimension, BlockDimension >> > (d_A, d_B, d_C);

	hipMemcpy(h_C, d_C, sizeof(int) * hc * wc, hipMemcpyDeviceToHost);
	for (int i = 0; i < hc; i++)
	{
		for (int j = 0; j < wc; j++)
		{
			printf("%d\t", h_C[i][j]);
		}
		printf("\n");
	}
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}